#include "cudaHelpers.cuh"

void cudaAssert(hipError_t code, const char *file, int line, bool abort)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "CUDAassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}
void cufftAssert(hipfftResult_t code, const char *file, int line, bool abort)
{
	if (code != HIPFFT_SUCCESS)
	{
		fprintf(stderr, "CUFFTassert: %s %s %d\n", _cudaGetErrorEnum(code), file, line);
		if (abort) exit(code);
	}
}



#ifdef _CUFFT_H_
// cuFFT API errors
const char *_cudaGetErrorEnum(hipfftResult error)
{
	switch (error)
	{
	case HIPFFT_SUCCESS:
		return "HIPFFT_SUCCESS";

	case HIPFFT_INVALID_PLAN:
		return "HIPFFT_INVALID_PLAN";

	case HIPFFT_ALLOC_FAILED:
		return "HIPFFT_ALLOC_FAILED";

	case HIPFFT_INVALID_TYPE:
		return "HIPFFT_INVALID_TYPE";

	case HIPFFT_INVALID_VALUE:
		return "HIPFFT_INVALID_VALUE";

	case HIPFFT_INTERNAL_ERROR:
		return "HIPFFT_INTERNAL_ERROR";

	case HIPFFT_EXEC_FAILED:
		return "HIPFFT_EXEC_FAILED";

	case HIPFFT_SETUP_FAILED:
		return "HIPFFT_SETUP_FAILED";

	case HIPFFT_INVALID_SIZE:
		return "HIPFFT_INVALID_SIZE";

	case HIPFFT_UNALIGNED_DATA:
		return "HIPFFT_UNALIGNED_DATA";
	}

	return "<unknown>";
}


#endif
__device__ float d_Lerp(float a, float b, float x)
{
	return a + (b - a) * x;
}
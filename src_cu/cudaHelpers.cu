#include "hip/hip_runtime.h"
#include "cudaHelpers.cuh"
#include "Generics.cuh"
void cudaAssert(hipError_t code, const char *file, int line, bool abort)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "CUDAassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}
void cufftAssert(hipfftResult_t code, const char *file, int line, bool abort)
{
	if (code != HIPFFT_SUCCESS)
	{
		fprintf(stderr, "CUFFTassert: %s %s %d\n", _cudaGetErrorEnum(code), file, line);
		if (abort) exit(code);
	}
}


__global__ void OwnComplexDivideByVectorKernel(tcomplex* d_input, tfloat* d_divisors, tcomplex* d_output, size_t elements);

__global__ void OwnComplexDivideSafeByVectorKernel(tcomplex* d_input, tfloat* d_divisors, tcomplex* d_output, size_t elements)
{
	d_input += elements * blockIdx.y;
	d_output += elements * blockIdx.y;

	tfloat val;
	for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
		id < elements;
		id += blockDim.x * gridDim.x)
	{
		val = d_divisors[id];
		if (abs(val) < 1e-15)
			val = 0;
		else
			val = (tfloat)1 / val;
		d_output[id] = d_input[id] * val;
	}
}

void d_OwnComplexDivideSafeByVector(tcomplex* d_input, tfloat* d_divisors, tcomplex* d_output, size_t elements, int batch)
{
	size_t TpB = tmin((size_t)256, elements);
	dim3 grid = dim3(tmin((elements + TpB - 1) / TpB, (size_t)32768), batch);
	OwnComplexDivideSafeByVectorKernel << <grid, (uint)TpB >> > (d_input, d_divisors, d_output, elements);
}

#ifdef _CUFFT_H_
// cuFFT API errors
const char *_cudaGetErrorEnum(hipfftResult error)
{
	switch (error)
	{
	case HIPFFT_SUCCESS:
		return "HIPFFT_SUCCESS";

	case HIPFFT_INVALID_PLAN:
		return "HIPFFT_INVALID_PLAN";

	case HIPFFT_ALLOC_FAILED:
		return "HIPFFT_ALLOC_FAILED";

	case HIPFFT_INVALID_TYPE:
		return "HIPFFT_INVALID_TYPE";

	case HIPFFT_INVALID_VALUE:
		return "HIPFFT_INVALID_VALUE";

	case HIPFFT_INTERNAL_ERROR:
		return "HIPFFT_INTERNAL_ERROR";

	case HIPFFT_EXEC_FAILED:
		return "HIPFFT_EXEC_FAILED";

	case HIPFFT_SETUP_FAILED:
		return "HIPFFT_SETUP_FAILED";

	case HIPFFT_INVALID_SIZE:
		return "HIPFFT_INVALID_SIZE";

	case HIPFFT_UNALIGNED_DATA:
		return "HIPFFT_UNALIGNED_DATA";
	}

	return "<unknown>";
}


#endif
__device__ float d_Lerp(float a, float b, float x)
{
	return a + (b - a) * x;
}

hipfftHandle d_OwnIFFTC2RGetPlan(int const ndimensions, int3 const dimensions, int batch);

void d_OwnIFFTC2R(tcomplex* const d_input, tfloat* const d_output, hipfftHandle* plan, int3 dimensions, int batch)
{
#ifdef GTOM_DOUBLE
	hipfftExecZ2D(*plan, d_input, d_output);
#else
	cufftErrchk(hipfftExecC2R(*plan, d_input, d_output));
#endif

	gtom::d_MultiplyByScalar(d_output, d_output, Elements(dimensions) * batch, 1.0f / (float)Elements(dimensions));
}

void d_OwnIFFTC2R(tcomplex* const d_input, tfloat* const d_output, hipfftHandle* plan)
{
#ifdef GTOM_DOUBLE
	hipfftExecZ2D(*plan, d_input, d_output);
#else
	cufftErrchk(hipfftExecC2R(*plan, d_input, d_output));
#endif
}

void d_OwnIFFTC2R(tcomplex* const d_input, tfloat* const d_output, int const ndimensions, int3 const dimensions, int batch, bool renormalize)
{
	hipfftHandle plan = d_OwnIFFTC2RGetPlan(ndimensions, dimensions, batch);
	if (renormalize)
		d_OwnIFFTC2R(d_input, d_output, &plan, dimensions, batch);
	else
		d_OwnIFFTC2R(d_input, d_output, &plan);
	cufftErrchk(hipfftDestroy(plan));
}

hipfftHandle d_OwnIFFTC2RGetPlan(int const ndimensions, int3 const dimensions, int batch)
{
	hipfftHandle plan;
	hipfftType direction = IS_TFLOAT_DOUBLE ? HIPFFT_Z2D : HIPFFT_C2R;
	int n[3] = { dimensions.z, dimensions.y, dimensions.x };

	cufftErrchk(hipfftPlanMany(&plan, ndimensions, n + (3 - ndimensions),
		NULL, 1, 0,
		NULL, 1, 0,
		direction, batch));

	//cufftSetCompatibilityMode(plan, CUFFT_COMPATIBILITY_NATIVE);

	return plan;
}
__global__ void MaxOpKernel(float* d_input1, float d_input2, float* d_output, size_t elements);
__global__ void SphereMaskFTKernel(tcomplex* d_input, tcomplex* d_output, int3 dims, int radius2);

void d_SphereMaskFT(tcomplex* d_input, tcomplex* d_output, int3 dims, int radius, uint batch)
{
	int TpB = tmin(128, NextMultipleOf(dims.x, 32));
	dim3 grid = dim3(dims.y, dims.z, batch);
	SphereMaskFTKernel << <grid, TpB >> > (d_input, d_output, dims, radius * radius);
}

__global__ void SphereMaskFTKernel(tcomplex* d_input, tcomplex* d_output, int3 dims, int radius2)
{
	int z = blockIdx.y;
	int y = blockIdx.x;

	d_input += blockIdx.z * ElementsFFT(dims) + (z * dims.y + y) * (dims.x / 2 + 1);
	d_output += blockIdx.z * ElementsFFT(dims) + (z * dims.y + y) * (dims.x / 2 + 1);

	int zp = z < dims.z / 2 + 1 ? z : z - dims.x;
	zp *= zp;
	int yp = y < dims.y / 2 + 1 ? y : y - dims.x;
	yp *= yp;

	for (int x = threadIdx.x; x < dims.x / 2 + 1; x += blockDim.x)
	{
		int r = x * x + yp + zp;

		if (r < radius2)
			d_output[x] = d_input[x];
		else
			d_output[x] = { 0, 0 };
	}
}

void d_MaxOp(float* d_input1, float input2, float* d_output, size_t elements)
{
	size_t TpB = tmin((size_t)256, elements);
	size_t totalblocks = tmin((elements + TpB - 1) / TpB, (size_t)8192);
	dim3 grid = dim3((uint)totalblocks);
	MaxOpKernel << <grid, (uint)TpB >> > (d_input1, input2, d_output, elements);
}

 __global__ void MaxOpKernel(float* d_input1, float d_input2, float* d_output, size_t elements)
{
	for (size_t id = blockIdx.x * blockDim.x + threadIdx.x;
		id < elements;
		id += blockDim.x * gridDim.x)
		d_output[id] = fmaxf(d_input1[id], d_input2);
}
